#include "hip/hip_runtime.h"
#include "decode_gpu.hpp"
#include "defs.hpp"
#include "marker.hpp"
#include "reader.hpp"
#include "util.hpp"

#include <cub/device/device_reduce.cuh>
#include <cub/device/device_scan.cuh>
#include <hip/hip_runtime.h>
#include <jpeggpu/jpeggpu.h>

#include <cassert>
#include <type_traits>
#include <vector>

namespace {

/// TODO remove these assumptions
/// Assumption 1 (ass-1): all non-luma is subsampled with the same factor
/// Assumption 2 (ass-2): Huffman table mapping

/// \brief "s", subsequence size in 32 bits. Paper uses 4 or 32 depending on the quality of the
///   encoded image. TODO value of 4 seems to get the decoder stuck.
constexpr int chunk_size       = 8; ///< "s", in 32 bits
constexpr int subsequence_size = chunk_size * 32; ///< size in bits
// subsequence size is in bits, it makes it easier if it is a multiple of eight for data reading
static_assert(subsequence_size % 8 == 0);

/// \brief Contains all required information about the last syncrhonization point for the
///   subsequence.
struct subsequence_info {
    /// \brief Bit(!) position in scan. "Location of the last detected codeword."
    ///   TODO size_t?
    int p;
    /// \brief The number of decoded symbols.
    int n;
    /// \brief The data unit index in the MCU. With the sampling factors, the color component
    ///   can be inferred. The paper calls this field "the current color component",
    ///   but merely checking the color component will not suffice.
    int c;
    /// \brief Zig-zag index.
    int z;
};

struct reader_state {
    const uint8_t* data;
    const uint8_t* data_end;
    int32_t cache; // new bits are at the least significant positions
    int cache_num_bits;
};

/// \brief Loads the next eight bits.
__device__ void load_byte(reader_state& rstate)
{
    assert(rstate.data < rstate.data_end);
    assert(rstate.cache_num_bits + 8 < 32);

    // byte stuffing and restart markers are removed beforehand, padding in front
    //   of restart markers is not
    const uint8_t next_byte = *(rstate.data++);
    rstate.cache            = (rstate.cache << 8) | next_byte;
    rstate.cache_num_bits += 8;
}

/// \brief If there are enough bits in the input stream, loads `num_bits` into cache.
__device__ void load_bits(reader_state& rstate, int num_bits)
{
    while (rstate.cache_num_bits < num_bits) {
        if (rstate.data >= rstate.data_end) {
            return; // no more data to load
        }

        load_byte(rstate);
    }
}

/// \brief Peeks `num_bits` from cache, does not remove them.
///   Assumes enough bits are present.
__device__ int select_bits(reader_state& rstate, int num_bits)
{
    assert(num_bits < 31);
    assert(rstate.cache_num_bits >= num_bits);

    // upper bits are zero
    return rstate.cache >> (rstate.cache_num_bits - num_bits);
}

/// \brief Removes `num_bits` from cache.
__device__ void discard_bits(reader_state& rstate, int num_bits)
{
    assert(rstate.cache_num_bits >= num_bits);
    // set discarded bits to zero (upper bits in the cache)
    rstate.cache = rstate.cache & ((1 << (rstate.cache_num_bits - num_bits)) - 1);
    rstate.cache_num_bits -= num_bits;
}

/// \brief Get the Huffman category from stream.
///
/// \tparam do_discard Whether to discard the bits that were read in the process.
/// \param[out] length Number of bits read.
template <bool do_discard = true>
uint8_t __device__ get_category(
    reader_state& rstate, int& length, const jpeggpu::huffman_table& table)
{
    load_bits(rstate, 16);

    // due to possibly guessing the huffman table wrong, there may not be enough bits left
    const int max_bits = min(rstate.cache_num_bits, 16);
    if (max_bits == 0) {
        // exit if there are no bits
        length = 0;
        return 0;
    }
    int i, code;
    for (i = 0; i < max_bits; ++i) {
        code                    = select_bits(rstate, i + 1);
        const bool is_last_iter = i == (max_bits - 1);
        if (code <= table.maxcode[i + 1] || is_last_iter) {
            break;
        }
    }
    assert(1 <= i + 1 && i + 1 <= 16);
    // termination condition: 1 <= i + 1 <= 16, i + 1 is number of bits
    if constexpr (do_discard) {
        discard_bits(rstate, i + 1);
    }
    length        = i + 1;
    const int idx = table.valptr[i + 1] + (code - table.mincode[i + 1]);
    if (idx < 0 || 256 <= idx) {
        // found a value that does not make sense. this can happen if the wrong huffman
        //   table is used. TODO is this the correct return value?
        return 0;
    }
    return table.huffval[idx];
}

__device__ int get_value(int num_bits, int code)
{
    return code < ((1 << num_bits) >> 1) ? (code + ((-1) << num_bits) + 1) : code;
}

/// \brief End of block, indicates all remaining coefficients in the data unit are zero.
constexpr int symbol_eob = INT32_MAX - 1;

__device__ void decode_next_symbol_dc(
    reader_state& rstate,
    int& length,
    int& symbol,
    int& run_length,
    const jpeggpu::huffman_table& table_dc,
    const jpeggpu::huffman_table& table_ac,
    int z)
{
    int category_length    = 0;
    const uint8_t category = get_category(rstate, category_length, table_dc);

    if (category != 0) {
        assert(0 < category && category < 17);
        load_bits(rstate, category);
        // there might not be `category` bits left
        if (rstate.cache_num_bits < category) {
            // eat all remaining so the `decode_subsequence` loop does not get stuck
            discard_bits(rstate, rstate.cache_num_bits);
            length     = category_length + rstate.cache_num_bits;
            symbol     = symbol_eob; // arbitrary symbol
            run_length = 0; // arbitrary length
            return;
        }
        const int offset = select_bits(rstate, category);
        discard_bits(rstate, category);
        const int value = get_value(category, offset);

        length = category_length + category;
        symbol = value;
    } else {
        length = category_length;
        symbol = 0;
    }

    // peek next to determine run (is always AC)
    {
        int len;
        const uint8_t s    = get_category<false>(rstate, len, table_ac);
        const int run      = (s >> 4);
        const int category = s & 0xf;

        if (category != 0) {
            run_length = run;
        } else {
            // either EOB or ZRL, which is treated as a symbol
            run_length = 0;
        }
    }
};

__device__ void decode_next_symbol_ac(
    reader_state& rstate,
    int& length,
    int& symbol,
    int& run_length,
    const jpeggpu::huffman_table& table,
    int z)
{
    int category_length = 0;
    // s = (run, category)
    const uint8_t s     = get_category(rstate, category_length, table);
    const int run       = (s >> 4);
    const int category  = s & 0xf;

    if (category != 0) {
        assert(0 < category && category < 17);
        load_bits(rstate, category);
        // there might not be `category` bits left
        if (rstate.cache_num_bits < category) {
            // eat all remaining so the `decode_subsequence` loop does not get stuck
            discard_bits(rstate, rstate.cache_num_bits);
            length     = category_length + rstate.cache_num_bits;
            symbol     = symbol_eob; // arbitrary symbol
            run_length = 0; // arbitrary length
            return;
        }
        const int offset = select_bits(rstate, category);
        discard_bits(rstate, category);
        const int value = get_value(category, offset);

        length = category_length + category;
        symbol = value;

        if (z + 1 <= 63) { // note: z already includes `run`
            // next value is ac coefficient, peek next to determine run
            {
                int len;
                const uint8_t s    = get_category<false>(rstate, len, table);
                const int run      = (s >> 4);
                const int category = s & 0xf;

                if (category != 0) {
                    run_length = run;
                } else {
                    // EOB or ZRL
                    run_length = 0;
                }
            }
        } else {
            // next table is dc
            run_length = 0;
        }
    } else {
        if (run == 15) {
            length     = category_length;
            symbol     = 0; // ZRL
            run_length = 15;

            if (z + 1 <= 63) {
                // there may be a symbol after the ZRL
                {
                    int len;
                    const uint8_t s    = get_category<false>(rstate, len, table);
                    const int run      = (s >> 4);
                    const int category = s & 0xf;

                    if (category != 0) {
                        run_length += run;
                    }
                }
            } else {
                // next is dc
            }
        } else {
            length     = category_length;
            symbol     = symbol_eob;
            run_length = 63 - z;
        }
    }
};

/// \brief Extracts coefficients from the bitstream while switching between DC and AC Huffman
/// tables.
///
/// - If symbol equals ZRL, 15 will be returned for run_length
/// - If symbol equals EOB, 63 - z will be returned for run_length, with z begin the current
///     index in the zig-zag sequence
///
/// \param[inout] rstate
/// \param[out] length The number of processed bits. Will be non-zero.
/// \param[out] symbol The decoded coefficient, provided the code was not EOB or ZRL.
/// \param[out] run_length The run-length of zeroes which the coefficient is followed by.
/// \param[in] table
/// \param[in] z Current index in the zig-zag sequence.
__device__ void decode_next_symbol(
    reader_state& rstate,
    int& length,
    int& symbol,
    int& run_length,
    const jpeggpu::huffman_table& table_dc,
    const jpeggpu::huffman_table& table_ac,
    int z)
{
    if (z == 0) {
        decode_next_symbol_dc(rstate, length, symbol, run_length, table_dc, table_ac, z);
    } else {
        decode_next_symbol_ac(rstate, length, symbol, run_length, table_ac, z);
    }
}

enum class component {
    y, // Y (YCbCR) or C (CMYK)
    cb, // Cb (YCbCR) or M (CMYK)
    cr, // Cr (YCbCR) or Y (CMYK)
    k // k (CMYK)
};

/// \brief Infer image components based on data unit index `c` (in MCU).
__device__ component calc_component(int ssx, int ssy, int c, int num_components)
{
    const int num_luma_data_units = ssx * ssy;

    if (c < num_luma_data_units) {
        return component::y;
    }

    assert(num_components > 1);

    if (c == num_luma_data_units) {
        return component::cb;
    }

    assert(num_components > 2);

    if (c == num_luma_data_units + 1) {
        return component::cr;
    }

    assert(num_components > 3);

    if (c == num_luma_data_units + 2) {
        return component::k;
    }

    assert(false);
}

struct const_state {
    const uint8_t* scan;
    const uint8_t* scan_end;
    const jpeggpu::huffman_table* table_luma_dc;
    const jpeggpu::huffman_table* table_luma_ac;
    const jpeggpu::huffman_table* table_chroma_dc;
    const jpeggpu::huffman_table* table_chroma_ac;
    int ssx;
    int ssy;
    int num_components;
    int num_data_units;
};

static_assert(std::is_trivially_copyable_v<const_state>);

/// \brief Algorithm 2.
///
/// \tparam is_overflow Whether `i` was decoded by another thread already. TODO word this better.
/// \tparam do_write Whether to write the coefficients to the output buffer.
template <bool is_overflow, bool do_write>
__device__ subsequence_info
decode_subsequence(int i, int16_t* out, subsequence_info* s_info, const_state cstate)
{
    subsequence_info info;
    info.p = i * subsequence_size; // start of i-th subsequence
    info.n = 0;
    info.c = 0; // start from the first data unit of the Y component
    info.z = 0;

    reader_state rstate;
    rstate.data           = cstate.scan + (info.p / 8); // subsequence_size is multiple of eight
    rstate.data_end       = cstate.scan_end;
    rstate.cache          = 0;
    rstate.cache_num_bits = 0;

    int position_in_output = 0;
    if constexpr (do_write) {
        position_in_output = s_info[i].n;
    }
    if constexpr (is_overflow) {
        // FIXME is this proper? if not doing this, an uninitialized read will occur due to not
        //   storing n in sync_intra
        info.p = s_info[i - 1].p;
        info.c = s_info[i - 1].c;
        info.z = s_info[i - 1].z;

        rstate.data        = cstate.scan + (info.p / 8);
        const int in_cache = (8 - (info.p % 8)) % 8; // bits still in cache
        if (in_cache > 0) {
            rstate.cache          = *(rstate.data++);
            rstate.cache_num_bits = 8;
            discard_bits(rstate, 8 - in_cache);
        }
    }

    subsequence_info last_symbol; // the last detected codeword
    const int scan_bit_size = (cstate.scan_end - cstate.scan) * 8;
    while (info.p < min((i + 1) * subsequence_size, scan_bit_size)) {
        const component comp =
            calc_component(cstate.ssx, cstate.ssy, info.c, cstate.num_components);
        int length     = 0;
        int symbol     = 0;
        int run_length = 0;
        // always returns length > 0 to ensure while loop does not get stuck
        decode_next_symbol(
            rstate,
            length,
            symbol,
            run_length,
            comp == component::y ? *cstate.table_luma_dc : *cstate.table_chroma_dc,
            comp == component::y ? *cstate.table_luma_ac : *cstate.table_chroma_ac,
            info.z);
        if (do_write) {
            // TODO could make a separate kernel for this
            out[position_in_output / 64 * 64 + jpeggpu::order_natural[position_in_output % 64]] =
                symbol == symbol_eob ? 0 : symbol;
        }
        if (do_write) {
            position_in_output += run_length + 1;
        }
        info.p += length;
        info.n += run_length + 1;
        info.z += run_length + 1;

        // TODO is EOB check needed?
        if (info.z >= 64 || symbol == symbol_eob) {
            // the data unit is complete
            info.z = 0;
            ++info.c;

            // ass-1
            const int num_data_units_in_mcu = cstate.ssx * cstate.ssy + (cstate.num_components - 1);
            if (info.c >= num_data_units_in_mcu) {
                // mcu is complete
                info.c = 0;
            }
        }

        last_symbol = info;

        // check if we have all blocks. this is needed since the scan is padded to a 8-bit multiple
        //   this problem is excerbated by restart intevals, where padding occurs more frequently
        if (position_in_output >= cstate.num_data_units * 64) {
            break;
        }
    }

    return last_symbol;
}

/// \brief Each thread handles one subsequence.
///   alg-3:05-23
///
/// \tparam block_size "b", the number of adjacent subsequences that form a sequence.
template <int block_size>
__global__ void sync_intra_sequence(
    subsequence_info* s_info, int num_subsequences, const_state cstate)
{
    assert(block_size == blockDim.x);
    const int bi = blockIdx.x;
    const int si = threadIdx.x;

    const int seq_global = bi * block_size;
    int subseq_global    = seq_global + si;

    if (subseq_global >= num_subsequences) {
        return;
    }

    bool synchronized = false;
    // paper uses `+ block_size` but `end` should be an index
    const int end     = min(seq_global + block_size - 1, num_subsequences - 1);
    // alg-3:10
    {
        subsequence_info info =
            decode_subsequence<false, false>(subseq_global, nullptr, s_info, cstate);
        s_info[subseq_global].p = info.p;
        // paper text does not mention `n` should be stored here, but if not storing `n`
        //   the first (of block) subsequence info's `n` will not be initialized
        s_info[subseq_global].n = info.n;
        s_info[subseq_global].c = info.c;
        s_info[subseq_global].z = info.z;
    }
    __syncthreads(); // wait until data of next subsequence is available
    ++subseq_global;
    while (!synchronized && subseq_global <= end) {
        subsequence_info info =
            decode_subsequence<true, false>(subseq_global, nullptr, s_info, cstate);
        if (info.p == s_info[subseq_global].p && info.c == s_info[subseq_global].c &&
            info.z == s_info[subseq_global].z) {
            // the decoding process of this thread has found the same "outcome" for the
            //   `subseq_global`th subsequence as the thread before it
            synchronized = true;
        }
        // FIXME inserted a sync, s_info[subseq_global] may be read in another thread's
        //   decode_subsequence
        __syncthreads();
        s_info[subseq_global] = info;
        ++subseq_global;
        __syncthreads();
    }
}

/// \brief Each thread handles one sequence, the last sequence is not handled.
template <int block_size>
__global__ void sync_inter_sequence(
    subsequence_info* s_info,
    int num_subsequences,
    const_state cstate,
    uint8_t* sequence_not_synced,
    int num_sequences)
{
    assert(blockIdx.x == 0); // required for syncing to work
    const int bi = threadIdx.x;
    if (bi >= num_sequences - 1) {
        return;
    }

    // last subsequence of sequence i
    int subseq_global = (bi + 1) * block_size;
    bool synchronized = false;
    // paper uses `+ block_size` but `end` should be an index
    const int end     = min(subseq_global + block_size - 1, num_subsequences - 1);
    while (!synchronized && subseq_global <= end) {
        subsequence_info info =
            decode_subsequence<true, false>(subseq_global, nullptr, s_info, cstate);
        if (info.p == s_info[subseq_global].p && info.c == s_info[subseq_global].c &&
            info.z == s_info[subseq_global].z) {
            // this means a synchronization point was found
            synchronized            = true;
            // TODO paper says bi - 1 but this will be 0 for the first thread?
            sequence_not_synced[bi] = false;
        }
        __syncthreads();
        s_info[subseq_global] = info; // FIXME paper gives no index
        ++subseq_global;
        __syncthreads();
    }
}

__global__ void decode_write(
    int16_t* out, subsequence_info* s_info, int num_subsequences, const_state cstate)
{
    const int si = blockIdx.x * blockDim.x + threadIdx.x;
    if (si >= num_subsequences) {
        return;
    }

    // only first thread does not do overflow
    constexpr bool do_write = true;
    if (si == 0) {
        decode_subsequence<false, do_write>(si, out, s_info, cstate);
    } else {
        decode_subsequence<true, do_write>(si, out, s_info, cstate);
    }
}

struct sum_subsequence_info {
    __device__ __forceinline__ subsequence_info
    operator()(const subsequence_info& a, const subsequence_info& b) const
    {
        // asserts in the comparison function are not great since CUB may execute the comparator on
        // garbage data if the block or warp is not completely full
        return {0, a.n + b.n, 0, 0};
    }
};

/// \brief Copy `subsequence_info::n` from `src` to `dst`.
__global__ void assign_sinfo_n(
    int num_subsequences, subsequence_info* dst, const subsequence_info* src)
{
    const int lid = blockDim.x * blockIdx.x + threadIdx.x;
    if (lid >= num_subsequences) {
        return;
    }

    assert(src[lid].n >= 0);
    dst[lid].n = src[lid].n;
}

} // namespace

bool jpeggpu::is_gpu_decode_possible(const jpeggpu::reader& reader)
{
    // not supported if non-luminance planes do not have the same subsampling
    //   this makes figuring the component out easier (c in s_info)
    for (int c = 1; c < reader.num_components; ++c) {
        if (reader.css.x[c] != 1 || reader.css.y[c] != 1) {
            return false;
        }
    }

    // ass-2
    if (reader.huff_map[0][jpeggpu::HUFF_DC] != 0 || reader.huff_map[0][jpeggpu::HUFF_AC] != 0) {
        return false;
    }
    for (int c = 1; c < reader.num_components; ++c) {
        if (reader.huff_map[c][jpeggpu::HUFF_DC] != 1 ||
            reader.huff_map[c][jpeggpu::HUFF_AC] != 1) {
            return false;
        }
    }

    return true;
}

size_t jpeggpu::calculate_gpu_decode_memory(const jpeggpu::reader& reader)
{
    size_t required = 0;
    // d_scan (can be less due to destuffing)
    required += gpu_alloc_size(reader.scan_size);
    // d_huff
    required += gpu_alloc_size(4 * sizeof(jpeggpu::huffman_table));
    // d_out
    size_t total_data_size = 0;
    for (int c = 0; c < reader.num_components; ++c) {
        total_data_size += reader.data_sizes_x[c] * reader.data_sizes_y[c];
    }
    required += gpu_alloc_size(total_data_size * sizeof(int16_t));
    // d_s_info
    const size_t scan_bit_size = reader.scan_size * 8;
    const int num_subsequences =
        ceiling_div(scan_bit_size, static_cast<unsigned int>(subsequence_size));
    required += gpu_alloc_size(num_subsequences * sizeof(subsequence_info));
    // d_sequence_not_synced
    constexpr int block_size = 256;
    const int num_sequences  = ceiling_div(num_subsequences, static_cast<unsigned int>(block_size));
    required += gpu_alloc_size((num_sequences - 1) * sizeof(uint8_t));
    // d_num_unsynced_sequence
    required += gpu_alloc_size(sizeof(int));
    // d_tmp_storage (reduction)
    size_t tmp_storage_bytes_reduction = 0;
    CHECK_CUDA(hipcub::DeviceReduce::Sum(
        nullptr,
        tmp_storage_bytes_reduction,
        reinterpret_cast<uint8_t*>(0),
        reinterpret_cast<int*>(0),
        num_sequences - 1,
        hipStreamDefault));
    required += gpu_alloc_size(tmp_storage_bytes_reduction);
    // d_reduce_out
    required += gpu_alloc_size(num_subsequences * sizeof(subsequence_info));
    // d_tmp_storage (scan)
    size_t tmp_storage_bytes_scan = 0;
    CHECK_CUDA(hipcub::DeviceScan::ExclusiveScan(
        nullptr,
        tmp_storage_bytes_scan,
        reinterpret_cast<subsequence_info*>(0),
        reinterpret_cast<subsequence_info*>(0),
        sum_subsequence_info{},
        subsequence_info{},
        num_subsequences,
        hipStreamDefault));

    return required;
}

#define CHECK_STAT(call)                                                                           \
    do {                                                                                           \
        jpeggpu_status stat = call;                                                                \
        if (stat != JPEGGPU_SUCCESS) {                                                             \
            return JPEGGPU_INTERNAL_ERROR;                                                         \
        }                                                                                          \
    } while (0)

jpeggpu_status jpeggpu::process_scan(
    jpeggpu::reader& reader,
    int16_t* (&d_image_qdct)[jpeggpu::max_comp_count],
    void*& d_tmp,
    size_t& tmp_size,
    hipStream_t stream)
{
    // destuff TODO GPU
    uint8_t* d_scan;
    int scan_size = 0;
    {
        std::vector<uint8_t> destuffed;
        destuffed.reserve(reader.scan_size);

        for (size_t i = 0; i < reader.scan_size; ++i) {
            const uint8_t byte = reader.scan_start[i];
            if (byte == 0xff) {
                assert(i + 1 < reader.scan_size);
                ++i;
                // skip next byte, check its value
                const uint8_t marker = reader.scan_start[i];
                // should be a stuffed byte or a restart marker
                assert(
                    marker == 0 ||
                    (jpeggpu::MARKER_RST0 <= marker && marker <= jpeggpu::MARKER_RST7));
                (void)marker; // surpress warning
                // stuffed byte or marker is subsequently ignored
            }
            destuffed.push_back(byte);
        }

        // this assumption allows to represent the bit offset with an int
        assert(destuffed.size() * 8 <= INT_MAX);

        scan_size = destuffed.size();

        CHECK_STAT(jpeggpu::gpu_alloc_reserve(
            reinterpret_cast<void**>(&d_scan), scan_size, d_tmp, tmp_size));
        CHECK_CUDA(hipMemcpy(d_scan, destuffed.data(), scan_size, hipMemcpyHostToDevice));
    }

    jpeggpu::huffman_table* d_huff;
    CHECK_STAT(jpeggpu::gpu_alloc_reserve(
        reinterpret_cast<void**>(&d_huff), 4 * sizeof(jpeggpu::huffman_table), d_tmp, tmp_size));
    CHECK_CUDA(hipMemcpyAsync(
        d_huff + 0,
        &(reader.huff_tables[0][jpeggpu::HUFF_DC]),
        sizeof(jpeggpu::huffman_table),
        hipMemcpyHostToDevice,
        stream));
    CHECK_CUDA(hipMemcpyAsync(
        d_huff + 1,
        &(reader.huff_tables[0][jpeggpu::HUFF_AC]),
        sizeof(jpeggpu::huffman_table),
        hipMemcpyHostToDevice,
        stream));
    CHECK_CUDA(hipMemcpyAsync(
        d_huff + 2,
        &(reader.huff_tables[1][jpeggpu::HUFF_DC]),
        sizeof(jpeggpu::huffman_table),
        hipMemcpyHostToDevice,
        stream));
    CHECK_CUDA(hipMemcpyAsync(
        d_huff + 3,
        &(reader.huff_tables[1][jpeggpu::HUFF_AC]),
        sizeof(jpeggpu::huffman_table),
        hipMemcpyHostToDevice,
        stream));

    const size_t scan_bit_size = scan_size * 8;
    const int num_subsequences =
        ceiling_div(scan_bit_size, static_cast<unsigned int>(subsequence_size)); // "N"
    constexpr int block_size = 256; // "b", size in subsequences
    const int num_sequences =
        ceiling_div(num_subsequences, static_cast<unsigned int>(block_size)); // "B"

    // alg-1:01
    size_t total_data_size = 0;
    int num_data_units     = 0;
    for (int c = 0; c < reader.num_components; ++c) {
        total_data_size += reader.data_sizes_x[c] * reader.data_sizes_y[c];
        num_data_units += (reader.data_sizes_x[c] / 8) * (reader.data_sizes_y[c] / 8);
    }
    int16_t* d_out;
    CHECK_STAT(jpeggpu::gpu_alloc_reserve(
        reinterpret_cast<void**>(&d_out), total_data_size * sizeof(int16_t), d_tmp, tmp_size));
    // initialize to zero, since only non-zeros are written
    CHECK_CUDA(hipMemsetAsync(d_out, 0, total_data_size * sizeof(int16_t), stream));

    // alg-1:05
    subsequence_info* d_s_info;
    CHECK_STAT(jpeggpu::gpu_alloc_reserve(
        reinterpret_cast<void**>(&d_s_info),
        num_subsequences * sizeof(subsequence_info),
        d_tmp,
        tmp_size));

    const const_state cstate = {
        d_scan,
        d_scan + scan_size,
        d_huff + 0,
        d_huff + 1,
        d_huff + 2,
        d_huff + 3,
        reader.css.x[0],
        reader.css.y[0],
        reader.num_components,
        num_data_units};

    { // sync_decoders (Algorithm 3)

        sync_intra_sequence<block_size>
            <<<num_sequences, block_size, 0, stream>>>(d_s_info, num_subsequences, cstate);
        CHECK_CUDA(hipGetLastError());

        if (num_sequences > 1) {
            // note: the meaning of this array is flipped, a one is produced if not synced
            uint8_t* d_sequence_not_synced;
            CHECK_STAT(jpeggpu::gpu_alloc_reserve(
                reinterpret_cast<void**>(&d_sequence_not_synced),
                (num_sequences - 1) * sizeof(uint8_t),
                d_tmp,
                tmp_size));
            CHECK_CUDA(hipMemsetAsync( // all are initialized to "not synced"
                d_sequence_not_synced,
                static_cast<uint8_t>(true),
                (num_sequences - 1) * sizeof(uint8_t),
                stream));

            int* d_num_unsynced_sequence;
            CHECK_STAT(jpeggpu::gpu_alloc_reserve(
                reinterpret_cast<void**>(&d_num_unsynced_sequence), sizeof(int), d_tmp, tmp_size));

            void* d_tmp_storage      = nullptr;
            size_t tmp_storage_bytes = 0;
            CHECK_CUDA(hipcub::DeviceReduce::Sum(
                d_tmp_storage,
                tmp_storage_bytes,
                d_sequence_not_synced,
                d_num_unsynced_sequence,
                num_sequences - 1,
                stream));

            CHECK_STAT(jpeggpu::gpu_alloc_reserve(
                reinterpret_cast<void**>(&d_tmp_storage), tmp_storage_bytes, d_tmp, tmp_size));

            int h_num_unsynced_sequence;
            do {
                // TODO this means the subsequence size must be dynamic.
                //   for the syncing in the kernel to work, only one block can be launched
                const int block_size_inter = num_sequences - 1;
                sync_inter_sequence<block_size><<<1, block_size_inter, 0, stream>>>(
                    d_s_info, num_subsequences, cstate, d_sequence_not_synced, num_sequences);
                CHECK_CUDA(hipDeviceSynchronize()); // FIXME remove
                std::cout << "inter sequence sync done\n";

                CHECK_CUDA(hipcub::DeviceReduce::Sum(
                    d_tmp_storage,
                    tmp_storage_bytes,
                    d_sequence_not_synced,
                    d_num_unsynced_sequence,
                    num_sequences - 1,
                    stream));
                CHECK_CUDA(hipMemcpy(
                    &h_num_unsynced_sequence,
                    d_num_unsynced_sequence,
                    sizeof(int),
                    hipMemcpyDeviceToHost));

                std::cout << "unsynced: " << h_num_unsynced_sequence << "\n";
            } while (h_num_unsynced_sequence);
        }
    }

    // FIXME debug
    std::vector<subsequence_info> h_s_info(num_subsequences);
    CHECK_CUDA(hipMemcpy(
        h_s_info.data(),
        d_s_info,
        num_subsequences * sizeof(subsequence_info),
        hipMemcpyDeviceToHost));

    // TODO consider SoA or do in-place
    // alg-1:07-08
    {
        subsequence_info* d_reduce_out;
        CHECK_STAT(jpeggpu::gpu_alloc_reserve(
            reinterpret_cast<void**>(&d_reduce_out),
            num_subsequences * sizeof(subsequence_info),
            d_tmp,
            tmp_size));
        // FIXME debug to satisfy initcheck
        CHECK_CUDA(
            hipMemsetAsync(d_reduce_out, 0, num_subsequences * sizeof(subsequence_info), stream));

        const subsequence_info init_value{0, 0, 0, 0};
        void* d_tmp_storage      = nullptr;
        size_t tmp_storage_bytes = 0;
        CHECK_CUDA(hipcub::DeviceScan::ExclusiveScan(
            d_tmp_storage,
            tmp_storage_bytes,
            d_s_info,
            d_reduce_out,
            sum_subsequence_info{},
            init_value,
            num_subsequences,
            stream));

        CHECK_STAT(jpeggpu::gpu_alloc_reserve(
            reinterpret_cast<void**>(&d_tmp_storage), tmp_storage_bytes, d_tmp, tmp_size));
        // FIXME debug to satisfy initcheck
        CHECK_CUDA(hipMemsetAsync(d_tmp_storage, 0, tmp_storage_bytes, stream));

        CHECK_CUDA(hipcub::DeviceScan::ExclusiveScan(
            d_tmp_storage,
            tmp_storage_bytes,
            d_s_info,
            d_reduce_out,
            sum_subsequence_info{},
            init_value,
            num_subsequences,
            stream));

        constexpr int block_size_assign = 256;
        const int grid_dim =
            ceiling_div(num_subsequences, static_cast<unsigned int>(block_size_assign));
        assign_sinfo_n<<<grid_dim, block_size_assign, 0, stream>>>(
            num_subsequences, d_s_info, d_reduce_out);
        CHECK_CUDA(hipGetLastError());
    }

    // FIXME debug
    CHECK_CUDA(hipMemcpy(
        h_s_info.data(),
        d_s_info,
        num_subsequences * sizeof(subsequence_info),
        hipMemcpyDeviceToHost));

    // alg-1:09-15
    CHECK_CUDA(hipStreamSynchronize(stream)); // FIXME debug
    decode_write<<<num_sequences, block_size, 0, stream>>>(
        d_out, d_s_info, num_subsequences, cstate);
    CHECK_CUDA(hipGetLastError());
    CHECK_CUDA(hipStreamSynchronize(stream)); // FIXME debug

    // FIXME can the below (and above) deal with non-interleaved scans?

    std::vector<std::vector<int16_t>> data(reader.num_components);
    for (int c = 0; c < reader.num_components; ++c) {
        data[c].resize(reader.data_sizes_x[c] * reader.data_sizes_y[c]);
    }

    // TODO replace with GPU transpose
    std::vector<int16_t> h_out(total_data_size);
    CHECK_CUDA(hipMemcpyAsync(
        h_out.data(), d_out, total_data_size * sizeof(int16_t), hipMemcpyDeviceToHost, stream));
    CHECK_CUDA(hipStreamSynchronize(stream));

    // TODO fix for non-interleaved

    int data_unit_idx = 0;
    for (int y = 0; y < reader.num_mcus_y; ++y) {
        for (int x = 0; x < reader.num_mcus_x; ++x) {
            constexpr size_t data_unit_bytes = jpeggpu::data_unit_size * sizeof(int16_t);
            for (int c = 0; c < reader.num_components; ++c) {
                for (int ssy = 0; ssy < reader.css.y[c]; ++ssy) {
                    for (int ssx = 0; ssx < reader.css.x[c]; ++ssx) {
                        const int y_block = y * reader.css.y[c] + ssy;
                        const int x_block = x * reader.css.x[c] + ssx;
                        const size_t idx  = y_block * jpeggpu::block_size * reader.mcu_sizes_x[c] *
                                               reader.num_mcus_x +
                                           x_block * jpeggpu::block_size * jpeggpu::block_size;
                        std::memcpy(
                            data[c].data() + idx,
                            h_out.data() + (data_unit_idx++) * jpeggpu::data_unit_size,
                            data_unit_bytes);
                    }
                }
            }
        }
    }

    // FIXME for subsampled images, it may be needed to first rearrange the data unit order
    //   for the luminance plane

    // undo DC difference encoding
    // TODO deal with non-interleaved?
    int dc[jpeggpu::max_comp_count] = {};
    int mcu_count                   = 0;
    for (int y_mcu = 0; y_mcu < reader.num_mcus_y; ++y_mcu) {
        for (int x_mcu = 0; x_mcu < reader.num_mcus_x; ++x_mcu) {
            if (reader.seen_dri && mcu_count % reader.restart_interval == 0) {
                for (int c = 0; c < jpeggpu::max_comp_count; ++c) {
                    dc[c] = 0;
                }
            }

            // one MCU
            for (int c = 0; c < reader.num_components; ++c) {
                for (int y_ss = 0; y_ss < reader.css.y[c]; ++y_ss) {
                    for (int x_ss = 0; x_ss < reader.css.x[c]; ++x_ss) {
                        const int y_block = y_mcu * reader.css.y[c] + y_ss;
                        const int x_block = x_mcu * reader.css.x[c] + x_ss;
                        const size_t idx  = y_block * jpeggpu::block_size * reader.mcu_sizes_x[c] *
                                               reader.num_mcus_x +
                                           x_block * jpeggpu::block_size * jpeggpu::block_size;
                        int16_t* dst = &(data[c][idx]);
                        dst[0]       = dc[c] += dst[0];
                    }
                }
            }
            ++mcu_count;
        }
    }

    for (int c = 0; c < reader.num_components; ++c) {
        CHECK_CUDA(hipMemcpy(
            d_image_qdct[c],
            data[c].data(),
            reader.data_sizes_x[c] * reader.data_sizes_y[c] * sizeof(int16_t),
            hipMemcpyHostToDevice));
    }

    return JPEGGPU_SUCCESS;
}

#undef CHECK_STAT
