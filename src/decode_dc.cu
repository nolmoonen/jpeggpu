// Copyright (c) 2024 Nol Moonen
//
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
//
// The above copyright notice and this permission notice shall be included in all
// copies or substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
// SOFTWARE.

#include "decode_dc.hpp"
#include "defs.hpp"
#include "logger.hpp"
#include "reader.hpp"

#include <jpeggpu/jpeggpu.h>

#include <cub/device/device_reduce.cuh>
#include <cub/device/device_scan.cuh>
#include <cub/thread/thread_operators.cuh>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/permutation_iterator.h>
#include <thrust/iterator/transform_iterator.h>

#include <hip/hip_runtime.h>

using namespace jpeggpu;

namespace {

struct interleaved_functor {
    interleaved_functor(int restart_interval, int data_units_in_mcu_component)
        : restart_interval(restart_interval),
          data_units_in_mcu_component(data_units_in_mcu_component)
    {
    }

    /// \brief For interleaved scans, returns segment index of data unit `i`.
    __device__ __host__ int operator()(int i)
    {
        const int num_data_units_in_segment = restart_interval * data_units_in_mcu_component;
        const int segment_idx               = i / num_data_units_in_segment;
        return segment_idx;
    }

    int restart_interval;
    int data_units_in_mcu_component;
};

struct interleaved_transform_functor {
    interleaved_transform_functor(
        int data_units_in_mcu_component, int off_in_mcu, int data_units_in_mcu)
        : data_units_in_mcu_component(data_units_in_mcu_component),
          off_in_mcu(off_in_mcu),
          data_units_in_mcu(data_units_in_mcu)
    {
    }

    /// \brief For interleaved scan, returns the pixel index of data unit `i`.
    __device__ __host__ int operator()(int i)
    {
        const int mcu_idx    = i / data_units_in_mcu_component;
        const int idx_in_mcu = off_in_mcu + i % data_units_in_mcu_component;

        const int data_unit_idx = mcu_idx * data_units_in_mcu + idx_in_mcu;
        const int data_idx      = data_unit_idx * data_unit_size;
        return data_idx;
    }

    int data_units_in_mcu_component;
    int off_in_mcu;
    int data_units_in_mcu;
};

} // namespace

template <bool do_it>
jpeggpu_status jpeggpu::decode_dc(
    const jpeg_stream& info,
    const scan& scan,
    int16_t* d_out,
    stack_allocator& allocator,
    hipStream_t stream,
    logger& logger)
{
    int off_in_mcu = 0; // number of data units, only used for interleaved

    for (int sc = 0; sc < scan.num_scan_components; ++sc) {
        const scan_component& scan_comp       = scan.scan_components[sc];
        const component& comp                 = info.components[scan_comp.component_idx];
        const int data_units_in_mcu_component = comp.ss.x * comp.ss.y;

        auto counting_iter = thrust::make_counting_iterator(int{0});

        // iterates over the DC values for the current component in interleaved scan
        auto interleaved_index_iter = thrust::make_transform_iterator(
            counting_iter,
            interleaved_transform_functor(
                data_units_in_mcu_component, off_in_mcu, scan.num_data_units_in_mcu));
        auto iter_interleaved = thrust::make_permutation_iterator(d_out, interleaved_index_iter);

        void* d_tmp_storage      = nullptr;
        size_t tmp_storage_bytes = 0;

        const int num_data_units_component =
            scan_comp.data_size.x * scan_comp.data_size.y / data_unit_size;

        if (info.restart_interval != 0) {
            // if restart interval is defined, scan by key where key is segment index

            auto counting_iter_key     = thrust::make_counting_iterator(int{0});
            const int restart_interval = info.restart_interval;
            auto iter_key              = thrust::make_transform_iterator(
                counting_iter_key,
                interleaved_functor(restart_interval, data_units_in_mcu_component));

            const auto dispatch = [&]() -> hipError_t {
                return hipcub::DeviceScan::InclusiveSumByKey(
                    d_tmp_storage,
                    tmp_storage_bytes,
                    iter_key,
                    iter_interleaved,
                    iter_interleaved,
                    num_data_units_component,
                    hipcub::Equality{},
                    stream);
            };

            JPEGGPU_CHECK_CUDA(dispatch());

            allocator.reserve<do_it>(&d_tmp_storage, tmp_storage_bytes);

            if (do_it) JPEGGPU_CHECK_CUDA(dispatch());
        } else {
            // if no restart interval is defined, simply perform a single scan

            const auto dispatch = [&]() -> hipError_t {
                return hipcub::DeviceScan::InclusiveSum(
                    d_tmp_storage,
                    tmp_storage_bytes,
                    iter_interleaved,
                    iter_interleaved,
                    num_data_units_component,
                    stream);
            };

            JPEGGPU_CHECK_CUDA(dispatch());

            allocator.reserve<do_it>(&d_tmp_storage, tmp_storage_bytes);

            if (do_it) JPEGGPU_CHECK_CUDA(dispatch());
        }

        off_in_mcu += data_units_in_mcu_component;
    }

    return JPEGGPU_SUCCESS;
}

template jpeggpu_status jpeggpu::decode_dc<false>(
    const jpeg_stream&, const scan&, int16_t*, stack_allocator&, hipStream_t, logger&);
template jpeggpu_status jpeggpu::decode_dc<true>(
    const jpeg_stream&, const scan&, int16_t*, stack_allocator&, hipStream_t, logger&);
