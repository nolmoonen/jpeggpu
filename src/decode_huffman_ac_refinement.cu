#include "hip/hip_runtime.h"
// Copyright (c) 2024 Nol Moonen
//
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
//
// The above copyright notice and this permission notice shall be included in all
// copies or substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
// SOFTWARE.

#include "decode_huffman_ac_refinement.hpp"
#include "decode_huffman_common.hpp"
#include "decode_huffman_reader.hpp"
#include "logger.hpp"
#include "reader.hpp"

using namespace jpeggpu;

namespace {

struct const_state {
    huffman_table* huff_tables;
    int16_t* out_0;
    int16_t* out_1;
    int16_t* out_2;
    int16_t* out_3;
    int num_data_units_0;
    int num_data_units_1;
    int num_data_units_2;
    int num_data_units_3;
};

struct tmp {
    uint8_t* scan_destuffed;
    segment* segments;
    int* segment_indices;
    int huff_idx;
    int comp_idx;
    int ss;
    int se;
    int al;
    int scan_size; // scan.end - scan.begin
};

template <int block_size>
__global__ void tmp2(tmp* tmps, const_state cstate)
{
    tmp t = tmps[blockIdx.x];

    __shared__ huffman_table table;
    load_huffman_table<block_size>(cstate.huff_tables[t.huff_idx], table);

    __shared__ uint32_t storage[block_size];
    // TODO making this explicit is better
    // this is safe because the scan allocation is a multiple of 256
    const int num_words_in_scan = ceiling_div(t.scan_size, 4u);
    if (threadIdx.x < num_words_in_scan) {
        storage[threadIdx.x] = reinterpret_cast<const uint32_t*>(t.scan_destuffed)[threadIdx.x];
    }
    assert(num_words_in_scan <= block_size); // FIXME remove once iterative load happens
    __syncthreads();

    if (threadIdx.x != 0) return;

    int16_t* out       = nullptr;
    int num_data_units = 0;
    switch (t.comp_idx) {
    case 0:
        out            = cstate.out_0;
        num_data_units = cstate.num_data_units_0;
        break;
    case 1:
        out            = cstate.out_1;
        num_data_units = cstate.num_data_units_1;
        break;
    case 2:
        out            = cstate.out_2;
        num_data_units = cstate.num_data_units_2;
        break;
    case 3:
        out            = cstate.out_3;
        num_data_units = cstate.num_data_units_3;
        break;
    }

    int word_idx      = 0;
    int bits_in_cache = 0; // can be zero if data has run out
    uint64_t cache    = 0;
    if (num_words_in_scan > 0) cache = uint64_t{storage[word_idx++]} << 32;
    if (num_words_in_scan > 1) cache |= storage[word_idx++];

    for (int i = 0; i < num_data_units; ++i) {
        const int se_exclusive = t.se + 1;
        for (int j = t.ss; j < se_exclusive; ++j) {
            if (bits_in_cache < 32) {
                const uint32_t word = word_idx++ < num_words_in_scan ? storage[word_idx++] : 0;
                cache |= uint64_t{word} << (32 - bits_in_cache);
                bits_in_cache += 32;
            }

            // ------------- at most 16 bits
            int i;
            int32_t code;
            huffman_table::entry entry;
            for (i = 0; i < 16; ++i) {
                code = u32_select_bits(static_cast<uint32_t>(cache >> 32), i + 1);
                const bool is_last_iter = i == 15;
                entry                   = table.entries[i];
                if (code <= entry.maxcode || is_last_iter) {
                    break;
                }
            }
            assert(1 <= i + 1 && i + 1 <= 16);
            // termination condition: 1 <= i + 1 <= 16, i + 1 is number of bits
            const int num_bits_consumed = i + 1;

            assert(num_bits_consumed <= bits_in_cache); // TODO make function
            bits_in_cache -= num_bits_consumed;
            cache <<= num_bits_consumed;

            const int idx = entry.valptr + (code - entry.mincode);
            assert(0 <= idx && idx < 256);
            const uint8_t s    = table.huffval[idx];
            const int run      = s >> 4;
            const int category = s & 0xf;

            const int positive = 1 << t.al;
            const int negative = ((unsigned)-1) << t.al; // TODO undefined behavior?

            if (category == 0 && run != 15) {
                // ---------------------- at most 14 bits
                // End of Band
                // read the next `run` bits (at most 14), contains #eob blocks
                const uint32_t eob_field = u32_select_bits(static_cast<uint32_t>(cache >> 32), run);

                assert(run <= bits_in_cache); // TODO make function
                bits_in_cache -= run;
                cache <<= run;

                // -1 to exclude the current block
                const int num_eob_blocks = eob_field + (uint32_t{1} << run) - 1;
                i += num_eob_blocks;

                // skip through all remaining
                for (; j < se_exclusive; ++j) {
                    const int coef_idx    = 64 * i + j;
                    const bool is_nonzero = out[coef_idx];
                    if (is_nonzero) {
                        const int code = u32_select_bits(static_cast<uint32_t>(cache >> 32), 1);

                        assert(1 <= bits_in_cache); // TODO make function
                        bits_in_cache -= 1;
                        cache <<= 1;

                        if (code) {
                            if ((out[coef_idx] & positive) == 0) {
                                if (out[coef_idx] >= 0) {
                                    out[coef_idx] += positive;
                                } else {
                                    out[coef_idx] += negative;
                                }
                            }
                        }
                    }
                }
                continue;
            }

            int coeff;
            if (category != 0) { // if not taking this branch, run == 15 aka ZRL
                assert(category == 1);
                const int code = u32_select_bits(static_cast<uint32_t>(cache >> 32), 1);

                assert(1 <= bits_in_cache); // TODO make function
                bits_in_cache -= 1;
                cache <<= 1;

                coeff = code ? positive : negative;
            }

            // -------------------- at most 62 bits
            int num_zeroes = run + 1;
            for (; j < se_exclusive; ++j) {
                const int coef_idx    = 64 * i + j;
                const bool is_nonzero = out[coef_idx];
                if (is_nonzero) {
                    const int code = u32_select_bits(static_cast<uint32_t>(cache >> 32), 1);

                    assert(1 <= bits_in_cache); // TODO make function
                    bits_in_cache -= 1;
                    cache <<= 1;

                    if (code) {
                        if ((out[coef_idx] & positive) == 0) {
                            if (out[coef_idx] >= 0) {
                                out[coef_idx] += positive;
                            } else {
                                out[coef_idx] += negative;
                            }
                        }
                    }
                } else {
                    --num_zeroes;
                    if (num_zeroes == 0) break;
                }
            }

            if (category != 0) {
                const int coef_idx = 64 * i + j;
                out[coef_idx]      = coeff;
            }
        }
    }
}

} // namespace

template <bool do_it>
jpeggpu_status jpeggpu::decode_ac_refinement(
    const jpeg_stream& info,
    const ac_scan_pass& scan_pass,
    uint8_t* (&d_scan_destuffed)[ac_scan_pass::max_num_scans],
    const std::vector<segment*>& d_segments,
    int* (&d_segment_indices)[ac_scan_pass::max_num_scans],
    int16_t* (&d_out)[max_comp_count],
    huffman_table* d_huff_tables,
    stack_allocator& allocator,
    hipStream_t stream,
    logger& logger)
{
    // FIXME deal with segments! probably can edit the loop to discard the remaining bits
    //   maybe destuffing code needs to be changed.

    tmp* d_tmps;
    JPEGGPU_CHECK_STAT(allocator.reserve<do_it>(&d_tmps, scan_pass.num_scans * sizeof(tmp)));

    if (do_it) {
        std::vector<tmp> h_tmps;
        JPEGGPU_CHECK_STAT(nothrow_resize(h_tmps, scan_pass.num_scans));
        for (int i = 0; i < scan_pass.num_scans; ++i) {
            const scan& scan                = info.scans[scan_pass.scan_indices[i]];
            const scan_component& scan_comp = scan.scan_components[0];
            tmp& t                          = h_tmps[i];
            t.scan_destuffed                = d_scan_destuffed[i];
            t.segments                      = d_segments[i];
            t.segment_indices               = d_segment_indices[i];
            t.huff_idx                      = scan_comp.ac_idx;
            t.comp_idx                      = scan_comp.component_idx;
            t.ss                            = scan.spectral_start;
            t.se                            = scan.spectral_end;
            t.al                            = scan.successive_approx_lo;
            // TODO what are the implications of this? see note in decode_huffman.cu
            t.scan_size = scan.end - scan.begin;
        }

        JPEGGPU_CHECK_CUDA(hipMemcpyAsync( // FIXME remove copy
            d_tmps,
            h_tmps.data(),
            scan_pass.num_scans * sizeof(tmp),
            hipMemcpyHostToDevice,
            stream));

        // TODO remove assert once variable is checked
        assert((info.components[0].size.x * info.components[0].size.y) % 64 == 0);
        assert((info.components[1].size.x * info.components[1].size.y) % 64 == 0);
        assert((info.components[2].size.x * info.components[2].size.y) % 64 == 0);
        assert((info.components[3].size.x * info.components[3].size.y) % 64 == 0);
        const const_state cstate = {
            d_huff_tables,
            d_out[0],
            d_out[1],
            d_out[2],
            d_out[3],
            info.components[0].size.x * info.components[0].size.y / 64,
            info.components[1].size.x * info.components[1].size.y / 64,
            info.components[2].size.x * info.components[2].size.y / 64,
            info.components[3].size.x * info.components[3].size.y / 64};

        constexpr int block_size = 256;
        tmp2<block_size><<<scan_pass.num_scans, block_size, 0, stream>>>(d_tmps, cstate);
        JPEGGPU_CHECK_CUDA(hipGetLastError());
    }

    return JPEGGPU_SUCCESS;
}

template jpeggpu_status jpeggpu::decode_ac_refinement<false>(
    const jpeg_stream&,
    const ac_scan_pass&,
    uint8_t* (&)[ac_scan_pass::max_num_scans],
    const std::vector<segment*>&,
    int* (&)[ac_scan_pass::max_num_scans],
    int16_t* (&)[max_comp_count],
    huffman_table*,
    stack_allocator&,
    hipStream_t,
    logger&);

template jpeggpu_status jpeggpu::decode_ac_refinement<true>(
    const jpeg_stream&,
    const ac_scan_pass&,
    uint8_t* (&)[ac_scan_pass::max_num_scans],
    const std::vector<segment*>&,
    int* (&)[ac_scan_pass::max_num_scans],
    int16_t* (&)[max_comp_count],
    huffman_table*,
    stack_allocator&,
    hipStream_t,
    logger&);
